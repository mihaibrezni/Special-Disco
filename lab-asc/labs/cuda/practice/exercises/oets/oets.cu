#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEMENTS 16
#define BLOCK_SIZE 16

__device__ void swap(int *a, int *b) {
	int temp = *a;
	*a = *b;
	*b = temp;
}

// TODO 2: define parameters
__global__ void oddEvenTranspositionSort(int *data, const size_t n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < n; i++) {
		if ((i & 1) == 0) {  // Even phase
			// TODO 2: Compare and swap elements if thread id is even
			if ((tid & 1) == 0 && tid < (n - 1))
			{
				if (data[tid] > data[tid + 1])
				{
					swap(&data[tid], &data[tid + 1]);
				}
			}
		} else {           // Odd phase
			// TODO 3: Compare and swap elements if thread id is odd
			if ((tid & 1) == 1 && tid < (n - 1))
			{
				if (data[tid] > data[tid + 1])
				{
					swap(&data[tid], &data[tid + 1]);
				}
			}
		}
		// TODO 4: Sync threads
		__syncthreads();
	}
}

void generateData(int *data, int size) {
	srand(time(0));

	for (int i = 0; i < size; i++) {
		data[i] = rand() % 14 + 1;
	}
}

int main() {
	int *array = NULL;
	array = (int *)malloc(NUM_ELEMENTS * sizeof(int));
	generateData(array, NUM_ELEMENTS);

	printf("Original Array: ");
	for (int i = 0; i < NUM_ELEMENTS; i++) {
		printf("%d ", array[i]);
	}
	printf("\n");

	int *d_array;
	// TODO 0: Allocate device array and copy host elements to it
	hipMalloc(&d_array, NUM_ELEMENTS * sizeof(int));
	hipMemcpy(d_array, array, NUM_ELEMENTS * sizeof(int),
				hipMemcpyHostToDevice);

	// TODO 1: Calculate blocks_no and block_size
	int blocks_no = NUM_ELEMENTS / BLOCK_SIZE;
	oddEvenTranspositionSort<<<blocks_no, BLOCK_SIZE>>>(d_array, NUM_ELEMENTS);
	hipDeviceSynchronize();

	hipMemcpy(array, d_array, NUM_ELEMENTS * sizeof(int),
				hipMemcpyDeviceToHost);
	hipFree(d_array);

	printf("Sorted Array: ");
	for (int i = 0; i < NUM_ELEMENTS; i++) {
		printf("%d ", array[i]);
	}
	printf("\n");

	free(array);
	return 0;
}
