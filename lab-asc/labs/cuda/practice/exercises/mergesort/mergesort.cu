#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_ELEMENTS 16

__device__ void merge(int* arr, int* temp, int left, int middle, int right) {
  int i = left;
  int j = middle;
  int k = left;

  while (i < middle && j < right) {
    // TODO 1: Get the smaller element
  }

  while (i < middle) {
    // TODO 2: Copy any remaining elements from the left subarray
  }
  while (j < right) {
    // TODO 3: Copy any remaining elements from the right subarray
  }

  for (int x = left; x < right; x++) {
    // TODO 4: Copy the sorted elements
  }
}

__global__ void mergeSortGPU(int* arr, int* temp, int n, int width) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int left = tid * width;
  int middle = left + width / 2;
  int right = left + width;

  if (left < n && middle < n) {
    merge(arr, temp, left, middle, right);
  }
}

void generateData(int* data, int size) {
  srand(time(0));

  for (int i = 0; i < size; i++) {
    data[i] = rand() % 14 + 1;
  }
}

int main() {
  int* array = NULL;
  array = (int*)malloc(NUM_ELEMENTS * sizeof(int));
  generateData(array, NUM_ELEMENTS);

  printf("Original Array: ");
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    printf("%d ", array[i]);
  }
  printf("\n");

  int *d_array, *d_tmp;
  hipMalloc(&d_array, NUM_ELEMENTS * sizeof(int));
  hipMemcpy(d_array, array, NUM_ELEMENTS * sizeof(int),
             hipMemcpyHostToDevice);
  hipMalloc(&d_tmp, NUM_ELEMENTS * sizeof(int));

  int block_size = 256;
  size_t blocks_no = NUM_ELEMENTS / block_size;

  if (NUM_ELEMENTS % block_size) ++blocks_no;

  for (int width = 1; width < NUM_ELEMENTS; width *= 2) {
    // TODO 0: Call the mergeSortGPU kernel with the appropriate arguments
  }
  hipDeviceSynchronize();

  hipMemcpy(array, d_array, NUM_ELEMENTS * sizeof(int),
             hipMemcpyDeviceToHost);
  hipFree(d_array);

  printf("Sorted Array: ");
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    printf("%d ", array[i]);
  }
  printf("\n");

  return 0;
}
