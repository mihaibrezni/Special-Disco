#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define KERNEL_SIZE 5
#define KERNEL_RADIUS (KERNEL_SIZE / 2)
#define BLOCK_SIZE 16
#define WIDTH 512
#define HEIGHT 512

#define PI 3.14159265358979323846

// Gaussian Kernel (fixed size for simplicity)
__constant__ float d_kernel[KERNEL_SIZE];

// TODO
__global__ void gaussianBlur(const unsigned char* input, unsigned char* output,
                             int width, int height) {
  // TODO 1: Calculate x and y

  // TODO 2: Check if values are within the image boundaries

  // TODO 3: Calculate the thread index

  float sum = 0.0;
  float weightSum = 0.0;

  for (int ky = -KERNEL_RADIUS; ky <= KERNEL_RADIUS; ky++) {
    for (int kx = -KERNEL_RADIUS; kx <= KERNEL_RADIUS; kx++) {
      // TODO 4: Calculate dX and dY

      // TODO 5: Check if values are within the image boundaries
      // and calculate the weight and sum
    }
  }

  // TODO 6: Calculate and store the output value
}

void setGaussianKernel(float sigma) {
  float sum = 0.0;
  float s = 2.0 * sigma * sigma;
  float kernel[KERNEL_SIZE];

  for (int i = 0; i < KERNEL_SIZE; i++) {
    int x = i - KERNEL_RADIUS;
    kernel[i] = (exp(-(x * x) / s)) / (PI * s);
    sum += kernel[i];
  }

  for (int i = 0; i < KERNEL_SIZE; i++) {
    kernel[i] /= sum;
  }

  hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, KERNEL_SIZE * sizeof(float));
}

int main() {
  unsigned char* h_image = (unsigned char*)malloc(WIDTH * HEIGHT);
  unsigned char* h_blurredImage = (unsigned char*)malloc(WIDTH * HEIGHT);

  // Generate random grayscale image
  for (int i = 0; i < WIDTH * HEIGHT; i++) {
    h_image[i] = rand() % 256;
  }
  stbi_write_png("original.png", WIDTH, HEIGHT, 1, h_image, WIDTH);

  unsigned char* d_image;
  unsigned char* d_blurredImage;
  hipMalloc(&d_image, WIDTH * HEIGHT);
  hipMalloc(&d_blurredImage, WIDTH * HEIGHT);

  hipMemcpy(d_image, h_image, WIDTH * HEIGHT, hipMemcpyHostToDevice);

  setGaussianKernel(1.0);  // Set sigma for Gaussian Kernel

  dim3 blocks((WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

  // TODO 0: Call the kernel function

  hipMemcpy(h_blurredImage, d_blurredImage, WIDTH * HEIGHT,
             hipMemcpyDeviceToHost);

  stbi_write_png("blurred.png", WIDTH, HEIGHT, 1, h_blurredImage, WIDTH);

  free(h_image);
  free(h_blurredImage);
  hipFree(d_image);
  hipFree(d_blurredImage);

  return 0;
}
