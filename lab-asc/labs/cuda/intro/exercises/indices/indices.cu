#include "hip/hip_runtime.h"
#include <stdio.h>

#include "utils.h"

/**
 * ~TODO 3~
 * Modify the kernel below such as each element of the
 * array will be now equal to 0 if it is an even number
 * or 1, if it is an odd number
 */
__global__ void kernel_parity_id(int *a, int N) {}

/**
 * ~TODO 4~
 * Modify the kernel below such as each element will
 * be equal to the BLOCK ID this computation takes
 * place.
 */
__global__ void kernel_block_id(int *a, int N) {}

/**
 * ~TODO 5~
 * Modify the kernel below such as each element will
 * be equal to the THREAD ID this computation takes
 * place.
 */
__global__ void kernel_thread_id(int *a, int N) {}

int main(void) {
  int nDevices;

  // Get the number of CUDA-capable GPU(s)
  hipGetDeviceCount(&nDevices);

  /**
   * ~TODO 1~
   * For each device, show some details in the format below,
   * then set as active device the first one (assuming there
   * is at least CUDA-capable device). Pay attention to the
   * type of the fields in the hipDeviceProp_t structure.
   *
   * Device number: <i>
   *      Device name: <name>
   *      Total memory: <mem>
   *      Memory Clock Rate (KHz): <mcr>
   *      Memory Bus Width (bits): <mbw>
   *
   * Hint: look for hipGetDeviceProperties and hipSetDevice in
   * the Cuda Toolkit Documentation.
   */
  for (int i = 0; i < nDevices; ++i) {
  }

  /**
   * ~TODO 2~
   * With information from example_2.cu, allocate an array with
   * integers (where a[i] = i). Then, modify the three kernels
   * above and execute them using 4 blocks, each with 4 threads.
   *
   * You can use the fill_array(int *a, int n) function (from utils)
   * to fill your array as many times you want.
   *
   *  ~TODO 3~
   * Execute kernel_parity_id kernel and then copy from
   * the device to the host; call hipDeviceSynchronize()
   * after a kernel execution for safety purposes.
   */

  // Uncomment the line below to check your results
  // check_task_2(3, host_array);

  /**
   * ~TODO 4~
   * Execute kernel_block_id kernel and then copy from
   * the device to the host;
   */

  // Uncomment the line below to check your results
  // check_task_2(4, host_array);

  /**
   * ~TODO 5~
   * Execute kernel_thread_id kernel and then copy from
   * the device to the host;
   */

  // Uncomment the line below to check your results
  // check_task_2(5, host_array);

  // TODO 6: Free the memory

  return 0;
}