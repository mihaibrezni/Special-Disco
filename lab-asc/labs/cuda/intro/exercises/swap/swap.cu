
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define BUF_2M (2 * 1024 * 1024)
#define BUF_32M (32 * 1024 * 1024)

int main(void) {
  hipSetDevice(0);

  int *host_array_a = 0;
  int *host_array_b = 0;

  int *device_array_a = 0;
  int *device_array_b = 0;
  int *device_array_c = 0;

  // TODO 1: Allocate the host's arrays:
  // host_array_a => 32M
  // host_array_b => 32M

  // TODO 2: Allocate the host's arrays:
  // device_array_a => 32M
  // device_array_b => 32M
  // device_array_c => 2M

  // Check for allocation errors
  if (host_array_a == 0 || host_array_b == 0 || device_array_a == 0 ||
      device_array_b == 0 || device_array_c == 0) {
    printf("[*] Error!\n");
    return 1;
  }

  for (int i = 0; i < BUF_32M; ++i) {
    host_array_a[i] = i % 32;
    host_array_b[i] = i % 2;
  }

  printf("Before swap:\n");
  printf("a[i]\tb[i]\n");
  for (int i = 0; i < 10; ++i) {
    printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
  }

  // TODO 3: Copy from host to device

  // TODO 4: Swap the buffers (BUF_2M values each iteration)
  // Hint 1: device_array_c should be used as a temporary buffer
  // Hint 2: cudaMemcpy

  // TODO 5: Copy from device to host

  printf("\nAfter swap:\n");
  printf("a[i]\tb[i]\n");
  for (int i = 0; i < 10; ++i) {
    printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
  }

  // TODO 6: Free the memory

  return 0;
}
