#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays

    // TODO 2: Allocate the device's arrays

    // TODO 3: Check for allocation errors


    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = the size.

    // TODO 5: Copy the host's arrays to device

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid

    // TODO 7: Copy back the results and then uncomment the checking function


    //check_task_3(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory

    return 0;
}