
#include <hip/hip_runtime.h>
#include <stdio.h>

#define OPS_SCALE (2048)

#define KERNEL_OPS_COUNT (2 * OPS_SCALE)

__global__ void kernel_gflops(float *a, float *b) {
  int idx = threadIdx.x;

  a[idx] = b[idx];  // 1 LOAD + 1 STORE, 0 FP32 ops, 0 FP64 ops

  /**
   * ~TODO~
   * Measure FP32 GFlops and FP64 Gflops of the device
   * Try and achieve close to theoretical peak performance
   */
}

void fill_array_int(int *a, int N) {
  for (int i = 0; i < N; ++i) {
    a[i] = i;
  }
}

void fill_array_float(float *a, int N) {
  for (int i = 0; i < N; ++i) {
    a[i] = (float)i;
  }
}

void fill_array_random(float *a, int N) {
  for (int i = 0; i < N; ++i) {
    a[i] = (float)rand() / RAND_MAX;
  }
}

int main(void) {
  int nDevices;

  // Get the number of CUDA-capable GPU(s)
  hipGetDeviceCount(&nDevices);
  hipSetDevice(0);

  float *device_a = 0;
  float *device_b = 0;
  float *host_a = 0;
  float *host_b = 0;

  int size = OPS_SCALE * OPS_SCALE;

  // Arrays a and b are of size N * N
  host_a = (float *)malloc(size * sizeof(float));
  host_b = (float *)malloc(size * sizeof(float));
  hipMalloc((void **)&device_a, size * sizeof(float));
  hipMalloc((void **)&device_b, size * sizeof(float));

  if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
    printf("[HOST] Couldn't allocate memory\n");
    return 1;
  }

  // Populate array a randomly
  fill_array_random(host_a, size);
  hipMemcpy(device_a, host_a, size * sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, stop;

  /**
   * ~TODO~
   * Create two cuda events (start and stop)
   * by using the cudaEventCreate function.
   */
  kernel_gflops<<<size / 512, 512>>>(device_a, device_b);
  hipEventSynchronize(stop);

  float ms = 0;
  float seconds = ms / pow((float)10, 3);

  /**
   *
   * Set num_ops to the number of floating point operations
   * done in the kernel multiplied with the size of the matrix.
   */

  double num_ops = (double)KERNEL_OPS_COUNT * size;
  double gflops = (double)num_ops / seconds / 1e+9;
  printf("GFLOPS: %.2f\n", gflops);

  free(host_a);
  free(host_b);
  hipFree(device_a);
  hipFree(device_b);

  return 0;
}
