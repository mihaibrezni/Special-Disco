
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

// CUDA kernel for performing saxpy operation
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

// Function to get the current CPU timer in seconds
double myCPUTimer() {
  return static_cast<double>(std::chrono::high_resolution_clock::now()
                                 .time_since_epoch()
                                 .count()) /
         1e9;
}

int main() {
  int N = 1 << 20;  // Example size of vectors
  float *x, *y, *d_x, *d_y;

  // Allocate host memory
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  // Initialize x and y arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Allocate device memory
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  // Copy inputs to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  double t1, t2;

  // Start CPU timer
  t1 = myCPUTimer();

  // Launch saxpy kernel
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  // Stop CPU timer
  t2 = myCPUTimer();

  // Copy result back to host
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  // Print the time in seconds
  std::cout << "Time using CPU timer: " << (t2 - t1) << " seconds."
            << std::endl;

  // Cleanup
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}
