
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for performing saxpy operation
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
  int N = 1 << 20;  // Example size of vectors
  float *x, *y, *d_x, *d_y;

  // Allocate host memory
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  // Initialize x and y arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Allocate device memory
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  // Copy inputs to device
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start);

  // Launch saxpy kernel
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  // Record the stop event
  hipEventRecord(stop);

  // Copy result back to host
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Convert milliseconds to seconds and print
  std::cout << "Time using CUDA events: " << milliseconds / 1000.0f
            << " seconds." << std::endl;

  // Cleanup
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}
