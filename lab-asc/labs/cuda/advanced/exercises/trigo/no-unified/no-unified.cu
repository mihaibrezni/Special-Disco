#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../utils.h"

#define NUM_ELEM (16 * 1024 * 1024)

__global__ void kernel_compute(float *a, float *b, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_NoUnifiedMem(void) {
  float *device_a = 0;
  float *device_b = 0;
  float *host_a = 0;
  float *host_b = 0;

  host_a = (float *)malloc(NUM_ELEM * sizeof(float));
  host_b = (float *)malloc(NUM_ELEM * sizeof(float));

  // TODO-plain: Alocare memorie (GPU/VRAM)
  if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
    printf("[HOST] Couldn't allocate memory\n");
    return 1;
  }

  fill_array_random(host_b, NUM_ELEM);

  // TODO-plain: Copiere date host_b (CPU/RAM) => device_b (GPU/VRAM)

  kernel_compute<<<NUM_ELEM / 256, 256>>>(device_a, device_b, NUM_ELEM);

  // TODO-plain: Copiere device_a (GPU/VRAM) => date host_a (CPU/RAM)

  // TODO print first partial results

  free(host_a);
  free(host_b);

  // TODO-plain: Dealocare memorie (GPU/VRAM)

  return 0;
}

int main(void) { compute_NoUnifiedMem(); }
