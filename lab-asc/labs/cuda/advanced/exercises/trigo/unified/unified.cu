#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../utils.h"

#define NUM_ELEM (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
  float* host_a = 0;
  float* host_b = 0;

  // TODO-unified: Alocare memorie unificata

  if (host_a == 0 || host_b == 0) {
    printf("[HOST] Couldn't allocate memory\n");
    return 1;
  }

  fill_array_random(host_b, NUM_ELEM);

  kernel_compute<<<NUM_ELEM / 256, 256>>>(host_a, host_b, NUM_ELEM);

  // TODO print first partial results

  // TODO-unified: Dealocare memorie unificata

  return 0;
}

int main(void) { compute_UnifiedMem(); }
