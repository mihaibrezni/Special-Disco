
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_ELEM 8
#define NUM_THREADS 10

__global__ void concurrentRW(int *data) {
  // NUM_THREADS try to read and write at same location

  atomicAdd(&data[blockIdx.x], threadIdx.x);
  // Equivalent to
  // data[blockIdx.x] = data[blockIdx.x] + threadIdx.x;
}

int main(int argc, char *argv[]) {
  int *data = NULL;
  bool errorsDetected = false;

  hipMallocManaged(&data, NUM_ELEM * sizeof(*data));
  if (data == 0) {
    perror("[HOST] Couldn't allocate memory");
    return 1;
  }

  // init all elements to 0
  hipMemset(data, 0, NUM_ELEM);

  // launch kernel writes
  concurrentRW<<<NUM_ELEM, NUM_THREADS>>>(data);
  hipDeviceSynchronize();
  if (hipSuccess != hipGetLastError()) {
    return 1;
  }

  for (int i = 0; i < NUM_ELEM; i++) {
    printf("%d. %d\n", i, data[i]);
    if (data[i] != (NUM_THREADS * (NUM_THREADS - 1) / 2)) {
      errorsDetected = true;
    }
  }

  if (errorsDetected) {
    perror("Errors detected");
  } else {
    puts("OK");
  }

  return 0;
}
