#include <hip/hip_runtime.h>

#include <fstream>
#include <iostream>

__device__ int pivot(int *v, int left, int right) {
  int pivotValue = v[left];

  while (left < right) {
    while (v[right] >= pivotValue && left < right) right--;
    v[left] = v[right];

    while (v[left] <= pivotValue && left < right) left++;
    v[right] = v[left];
  }

  v[left] = pivotValue;
  return left;
}

__global__ void quicksort(int *v, int left, int right, hipStream_t parentStream) {
  if (left < right) {
    int pivotIndex = pivot(v, left, right); // Assume pivot is device function or inlined
    
    hipStream_t leftStream, rightStream;
    hipStreamCreateWithFlags(&leftStream, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&rightStream, hipStreamNonBlocking);

    if (left < pivotIndex) {
        quicksort<<<1, 1, 0, leftStream>>>(v, left, pivotIndex - 1, leftStream);
    }
    if (pivotIndex + 1 < right) {
        quicksort<<<1, 1, 0, rightStream>>>(v, pivotIndex + 1, right, rightStream);
    }

    hipStreamDestroy(leftStream);
    hipStreamDestroy(rightStream);
  }
}

int main() {
  int *v;  // Unified pointer
  int n;

  // Read from file
  std::ifstream fin("input.txt");
  if (!fin) {
    std::cerr << "Failed to open input file." << std::endl;
    return -1;
  }

  fin >> n;
  hipMallocManaged(&v, n * sizeof(int));
  if (v == NULL)
    return -1;

  for (int i = 0; i < n; i++)
    fin >> v[i];
  fin.close();

  hipStream_t stream;
  hipStreamCreate(&stream); // Create a non-blocking stream

  // Call the quicksort kernel with the newly created stream
  quicksort<<<1, 1, 0, stream>>>(v, 0, n - 1, stream);

  // Synchronize the stream to ensure sorting completes before proceeding
  hipStreamSynchronize(stream);

  // Cleanup
  hipStreamDestroy(stream);

  // Print sorted array
  for (int i = 0; i < n; i++)
    std::cout << v[i] << " ";
  std::cout << std::endl;

  hipFree(v);

  return 0;
}
